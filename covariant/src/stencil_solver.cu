#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hlmv.hpp>
#include <dlmv.hpp>
#include <hArray.hpp>
#include <dArray.hpp>
#include <dcsr.hpp>
#include <hcsr.hpp>
#include <dict.hpp>
#include <hipsparse.h>
#include <string>

#include "helper.hpp"
#include "collocated.cu"
#include "staggered.cu"

int main(int argc, char **argv) {


        if (argc != 2) {
                printf("usage: %s <project_dir> \n", argv[0]);
                return -1;
        }

        hipblasHandle_t cublasH;
        cublasErrCheck(hipblasCreate(&cublasH));


        std::string project_dir = std::string(argv[1]) + "/";

        printf("Project directory: %s \n", project_dir.c_str());
        Dict cfg = read(project_dir + "config.txt");
        int nt = 2000;
        int nx = cfg["nx"];
        int ny = cfg["ny"];
        int scheme = cfg["scheme"];
        int my = ny;
        int ninfo = 100;
        int nvtk = 100;
        Tv hi1 = cfg["hi1"];
        Tv hi2 = cfg["hi2"];
        Tv dt = cfg["dt"];
        dump(cfg);

        hArray<Tv> p_ = read(project_dir + "p.bin");
        hArray<Tv> v1_ = read(project_dir + "v1.bin");
        hArray<Tv> v2_ = read(project_dir + "v2.bin");

        // Metrics, collocated
        hArray<Tv> J_ = read(project_dir + "J.bin");
        hArray<Tv> g11_ = read(project_dir + "g11.bin");
        hArray<Tv> g12_ = read(project_dir + "g12.bin");
        hArray<Tv> g22_ = read(project_dir + "g22.bin");

        // Metrics, staggered
        hArray<Tv> Jp_ = read(project_dir + "Jp.bin");
        hArray<Tv> J1_ = read(project_dir + "J1.bin");
        hArray<Tv> J2_ = read(project_dir + "J2.bin");
        hArray<Tv> g1_11_ = read(project_dir + "g1_11.bin");
        hArray<Tv> gp_12_ = read(project_dir + "gp_12.bin");
        hArray<Tv> g2_22_ = read(project_dir + "g2_22.bin");

        hArray<Tv> xp_ = read(project_dir + "xp.bin");
        hArray<Tv> yp_ = read(project_dir + "yp.bin");
        hArray<Tv> d_ = read(project_dir + "d.bin");
        
        //Fields
        dArray<Tv> p = htod(p_);
        dArray<Tv> v1 = htod(v1_);
        dArray<Tv> v2 = htod(v2_);

        ////Rates
        dArray<Tv> dp(p.size);
        dArray<Tv> dv1(v1_.size);
        dArray<Tv> dv2(v2_.size);

        // Collocated Metrics
        dArray<Tv> J = htod(J_);
        dArray<Tv> g11 = htod(g11_);
        dArray<Tv> g12 = htod(g12_);
        dArray<Tv> g22 = htod(g22_);

        // Staggered Metrics
        dArray<Tv> Jp = htod(Jp_);
        dArray<Tv> J1 = htod(J1_);
        dArray<Tv> J2 = htod(J2_);
        dArray<Tv> g1_11 = htod(g1_11_);
        dArray<Tv> gp_12 = htod(gp_12_);
        dArray<Tv> g2_22 = htod(g2_22_);

        dArray<Tv> d = htod(d_);

        col_init(a, b);

        Tv t = 0.0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int vtk_step = 0;
        float elapsed = 0;
        hipEventRecord(start);
        printf("step \t time \t elapsed (s) \t time per step (ms) \n");
        int run = 1;

        if (run) {
        col_periodic(J, g11, g22, nx, ny, my);
        for (int step = 0; step < nt; ++step) {

                if ( ninfo > 0 && step % ninfo == 0) {
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);
                        hipEventElapsedTime(&elapsed, start, stop);
                        printf("%3d \t %3.4f \t %3.4f \t %3.4f \n", step, t,
                               elapsed * 1e-3, elapsed / step);
                }

                if ( nvtk > 0 && step % nvtk == 0) {
                        dtoh(p_, p);
                        std::string out = project_dir + "p_" +
                                          std::to_string(vtk_step) + ".vtk";
                        write_vtk(out, p_.x, xp_, yp_, nx, ny);
                        vtk_step++;
                }

                for (int k = 0; k < rk4_n; ++k) {
                        t = step * dt + c[k]*dt;

                        if (scheme == 0) {
                                col_periodic(p, v1, v2, nx, ny, my);
                                col_rates(dp, dv1, dv2, p, v1, v2, J, g11, g12,
                                          g22, nx, ny, my, hi1, hi2, k);
                        } else if (scheme == 1) {
                                st_periodic(p, v1, v2, nx, ny, my);
                                st_rates(dp, dv1, dv2, p, v1, v2, Jp, J1, J2,
                                                g1_11, gp_12,
                                         g2_22, nx, ny, my, hi1, hi2, k);

                        }

                        // dp = dp + d * g(t + c[k]*dt)
                        Tv gval = ricker(t);
                        axpy(cublasH, dp, d, gval);

                        if (scheme == 0) {
                                col_update(p, v1, v2, dp, dv1, dv2, nx, ny, my,
                                           dt, k);
                        } else if (scheme == 1) {
                                st_update(p, v1, v2, dp, dv1, dv2, nx, ny, my,
                                          dt, k);
                        }
                }
                t = (1 + step) * dt;

        }
        }  else {
        //dump2d(Jp_, nx, ny);
        //dump2d(J1_, nx, ny);
        //dump2d(J2_, nx, ny);
        //dump2d(g1_11_, nx, ny);
        //dump2d(gp_12_, nx, ny);
        //dump2d(g2_22_, nx, ny);
        dump2d(Jp_, nx, ny);
        dump2d(gp_12_, nx, ny);
        dump2d(p_, nx, ny);
        //Dump2d(v1_, nx, ny);
        //Dump2d(v2_, nx, ny);

        //col_periodic(p, v1, v2, nx, ny, my);
        st_rates(dp, dv1, dv2, p, v1, v2, Jp, J1, J2, g1_11, gp_12, g2_22, nx,
                                  ny, my, hi1, hi2, 0);

        dtoh(p_, dp);
        dtoh(v1_, dv1);
        dtoh(v2_, dv2);
        dump2d(v1_, nx, ny);
        dump2d(v2_, nx, ny);


        }
        //dump2d(v1_, nx, ny);
        //dump2d(v2_, nx, ny);

        //dump2d(g11_, nx, ny);
        //dump2d(g12_, nx, ny);
        //dump2d(g22_, nx, ny);

        cublasErrCheck(hipblasDestroy(cublasH));

        return 0;
}
