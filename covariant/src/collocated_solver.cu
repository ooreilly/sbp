#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hlmv.hpp>
#include <dlmv.hpp>
#include <hArray.hpp>
#include <dArray.hpp>
#include <dcsr.hpp>
#include <hcsr.hpp>
#include <dict.hpp>
#include <hipsparse.h>
#include <string>

typedef float Tv;
typedef int Ti;

#include "helper.hpp"
#include "collocated.cu"
#include "staggered.cu"

int main(int argc, char **argv) {


        if (argc != 2) {
                printf("usage: %s <project_dir> \n", argv[0]);
                return -1;
        }

        hipblasHandle_t cublasH;
        cublasErrCheck(hipblasCreate(&cublasH));


        std::string project_dir = std::string(argv[1]) + "/";

        printf("Project directory: %s \n", project_dir.c_str());
        Dict cfg = read(project_dir + "config.txt");
        int nt = cfg["nt"];
        int nx = cfg["nx"];
        int ny = cfg["ny"];
        int scheme = cfg["scheme"];
        int my = ny;
        int ninfo = 100;
        int nvtk = cfg["nvtk"];
        Tv hi1 = cfg["hi1"];
        Tv hi2 = cfg["hi2"];
        Tv dt = cfg["dt"];

        std::string output_dir = cfg["outputdir"];
        output_dir += "/";
        dump(cfg);

        hArray<Tv> p_ = read(project_dir + "p.bin");
        hArray<Tv> v1_ = read(project_dir + "v1.bin");
        hArray<Tv> v2_ = read(project_dir + "v2.bin");

        hArray<Tv> J_ = read(project_dir + "J.bin");
        hArray<Tv> g11_ = read(project_dir + "g11.bin");
        hArray<Tv> g12_ = read(project_dir + "g12.bin");
        hArray<Tv> g22_ = read(project_dir + "g22.bin");

        dArray<Tv> J = htod(J_);
        dArray<Tv> g11 = htod(g11_);
        dArray<Tv> g12 = htod(g12_);
        dArray<Tv> g22 = htod(g22_);

        hArray<Tv> xp_ = read(project_dir + "xp.bin");
        hArray<Tv> yp_ = read(project_dir + "yp.bin");
        hArray<Tv> d_ = read(project_dir + "d.bin");
        
        //Fields
        dArray<Tv> p = htod(p_);
        dArray<Tv> v1 = htod(v1_);
        dArray<Tv> v2 = htod(v2_);

        //Rates
        dArray<Tv> dp(p.size);
        dArray<Tv> dv1(v1_.size);
        dArray<Tv> dv2(v2_.size);


        dArray<Tv> d = htod(d_);

        col_init(a, b);

        Tv t = 0.0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int vtk_step = 0;
        float elapsed = 0;
        hipEventRecord(start);
        printf("step \t time \t elapsed (s) \t time per step (ms) \n");

        for (int step = 0; step < nt; ++step) {

                if ( ninfo > 0 && step % ninfo == 0) {
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);
                        hipEventElapsedTime(&elapsed, start, stop);
                        printf("%3d \t %3.4f \t %3.4f \t %3.4f \n", step, t,
                               elapsed * 1e-3, elapsed / step);
                }

                if ( nvtk > 0 && step % nvtk == 0) {
                        dtoh(p_, p);
                        std::string out = output_dir + "p_" +
                                          std::to_string(vtk_step) + ".vtk";
                        write_vtk(out, p_.x, xp_, yp_, nx, ny);
                        vtk_step++;
                }

                for (int k = 0; k < rk4_n; ++k) {
                        t = step * dt + c[k]*dt;

                        col_rates(dp, dv1, dv2, p, v1, v2, J, g11, g12, g22, nx,
                                  ny, my, hi1, hi2, k);

                        // dp = dp + d * g(t + c[k]*dt)
                        Tv gval = ricker(t);
                        axpy(cublasH, dp, d, gval);

                        col_update(p, v1, v2, dp, dv1, dv2, nx, ny, my, dt, k);
                }
                t = (1 + step) * dt;

        }
        
        // Write log file
        {
        std::string out = output_dir + "log.txt";
        FILE *fh = fopen(out.c_str(), "w");
        fprintf(fh, "nx=%d\n", nx);
        fprintf(fh, "ny=%d\n", ny);
        fprintf(fh, "elapsed=%g\n", elapsed * 1e-3);
        fprintf(fh, "nt=%d\n", nt);
        fclose(fh);
        }

        cublasErrCheck(hipblasDestroy(cublasH));

        return 0;
}
